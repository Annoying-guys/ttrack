#include "hip/hip_runtime.h"
#include "../../../../include/ttrack/track/localizer/levelsets/pwp3d_cuda.hpp"

__global__ void testCudaFunction(int *a, int *b, int *c){

  *c = *a + *b;
  //data[0] = 4;

}


bool ttrk::gpu::checkCudaFunctionality(){

  int ct;
  hipGetDeviceCount(&ct);
  if (ct == 0){
    return false;
  }

  hipError_t code = hipGetLastError();
  
  for (int dev = 0; dev < ct; ++dev){
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, dev);
  }

  return true;
  //
  //int a, b, c;
  //a = 4;
  //b = 6;

  //int *a_d, *b_d, *c_d;
  //hipMalloc((void **)&a_d, sizeof(int));
  //hipMalloc((void **)&b_d, sizeof(int));
  //hipMalloc((void **)&c_d, sizeof(int));
  //hipMemcpy(a_d, &a, sizeof(int), hipMemcpyHostToDevice);
  //hipMemcpy(b_d, &b, sizeof(int), hipMemcpyHostToDevice);

  //testCudaFunction<<<1,1>>>(a_d, b_d, c_d);
  //hipMemcpy(&c, c_d, sizeof(int), hipMemcpyDeviceToHost);
  //code = hipGetLastError();
  //
  //hipMemcpy(&c, c_d, sizeof(int), hipMemcpyDeviceToHost);
  //hipFree(a_d);
  //hipFree(b_d);
  //hipFree(c_d);
  ////ci::app::console() << "P = " << p << std::endl;
  //int j = 0;
  //int x = j + 3;

}